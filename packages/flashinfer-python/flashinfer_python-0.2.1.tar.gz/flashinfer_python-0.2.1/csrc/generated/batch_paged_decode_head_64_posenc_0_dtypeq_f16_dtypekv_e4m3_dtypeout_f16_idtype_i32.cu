#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = BatchDecodeParams<half, __hip_fp8_e4m3_fnuz, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);


using ParamsMlaT = BatchDecodeParamsMLA<half, __hip_fp8_e4m3_fnuz, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<64, 8, ComposedAttention<ParamsMlaT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsMlaT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);
}
    