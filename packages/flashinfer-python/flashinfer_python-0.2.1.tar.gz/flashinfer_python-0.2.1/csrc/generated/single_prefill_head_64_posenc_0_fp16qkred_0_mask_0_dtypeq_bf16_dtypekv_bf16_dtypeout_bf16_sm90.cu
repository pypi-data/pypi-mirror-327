#include "hip/hip_runtime.h"
 // single_prefill_sm90 template inst
#include <flashinfer/attention/hopper/params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>

namespace flashinfer {

using DTypeQ = cutlass_dtype_t<hip_bfloat16>;
using DTypeKV = cutlass_dtype_t<hip_bfloat16>;
using DTypeO = cutlass_dtype_t<hip_bfloat16>;

using Params = SinglePrefillParams<DTypeQ, DTypeKV, DTypeO>;

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, MaskMode::kNone, /*USE_SWA=*/true, LogitsSoftCap<Params>>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, MaskMode::kNone, /*USE_SWA=*/false, LogitsSoftCap<Params>>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, MaskMode::kNone, /*USE_SWA=*/true, StandardAttention<Params>>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, MaskMode::kNone, /*USE_SWA=*/false, StandardAttention<Params>>
    (Params& params, hipStream_t stream);

}
    