#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SingleDecodeParams<__hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz>;

template hipError_t SingleDecodeWithKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);
}
    