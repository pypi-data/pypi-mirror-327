#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SinglePrefillParams<hip_bfloat16, __hip_fp8_e5m2_fnuz, hip_bfloat16>;

template hipError_t SinglePrefillWithKVCacheDispatched<64, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    