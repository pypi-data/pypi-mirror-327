#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SinglePrefillParams<half, half, half>;

template hipError_t SinglePrefillWithKVCacheDispatched<64, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp,
    hipStream_t stream);

}
    