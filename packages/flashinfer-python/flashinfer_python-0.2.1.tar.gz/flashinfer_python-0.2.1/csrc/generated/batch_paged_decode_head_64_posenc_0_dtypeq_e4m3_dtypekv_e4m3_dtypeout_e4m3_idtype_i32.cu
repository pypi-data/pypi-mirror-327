#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = BatchDecodeParams<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);


using ParamsMlaT = BatchDecodeParamsMLA<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<64, 8, ComposedAttention<ParamsMlaT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsMlaT params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);
}
    