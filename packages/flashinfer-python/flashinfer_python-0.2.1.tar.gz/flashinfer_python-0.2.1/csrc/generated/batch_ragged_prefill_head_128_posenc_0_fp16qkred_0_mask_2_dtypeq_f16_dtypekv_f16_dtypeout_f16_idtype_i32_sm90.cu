 // batch_ragged_prefill_sm90 template inst
#include <flashinfer/attention/hopper/params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>


namespace flashinfer {

using DTypeQ = cutlass_dtype_t<half>;
using DTypeKV = cutlass_dtype_t<half>;
using DTypeO = cutlass_dtype_t<half>;

using Params = BatchPrefillRaggedParams<DTypeQ, DTypeKV, DTypeO, int32_t>;


template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap<Params>>(Params& params, hipStream_t stream);
        


template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention<Params>>(Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithRaggedKVCacheDispatched
    <128,
     MaskMode::kCustom,
     /*USE_SWA=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention<Params>>(Params& params, hipStream_t stream);
        

}
    