#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SingleDecodeParams<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz>;

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);
}
    