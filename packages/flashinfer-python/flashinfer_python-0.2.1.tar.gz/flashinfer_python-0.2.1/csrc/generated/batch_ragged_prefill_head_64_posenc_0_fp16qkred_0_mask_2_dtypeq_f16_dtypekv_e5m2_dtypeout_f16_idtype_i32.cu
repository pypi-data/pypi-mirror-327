#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = BatchPrefillRaggedParams<half, __hip_fp8_e5m2_fnuz, half, int32_t>;

using AttentionVariant1 = ComposedAttention<ParamsT, get_variant_code(
    true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant2 = ComposedAttention<ParamsT, get_variant_code(
    true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2>(
    ParamsT params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

}
    