#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = BatchDecodeParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);


using ParamsMlaT = BatchDecodeParamsMLA<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<64, 8, ComposedAttention<ParamsMlaT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsMlaT params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);
}
    