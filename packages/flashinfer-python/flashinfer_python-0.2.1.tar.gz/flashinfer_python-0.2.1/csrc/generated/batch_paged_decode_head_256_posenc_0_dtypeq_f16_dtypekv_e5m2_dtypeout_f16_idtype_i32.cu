#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = BatchDecodeParams<half, __hip_fp8_e5m2_fnuz, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);


using ParamsMlaT = BatchDecodeParamsMLA<half, __hip_fp8_e5m2_fnuz, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<256, 32, ComposedAttention<ParamsMlaT, get_variant_code(
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsMlaT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);
}
    