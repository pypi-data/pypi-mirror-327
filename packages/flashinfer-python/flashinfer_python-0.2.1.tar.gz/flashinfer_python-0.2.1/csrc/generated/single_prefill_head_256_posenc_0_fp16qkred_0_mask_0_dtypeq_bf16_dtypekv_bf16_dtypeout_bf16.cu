#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SinglePrefillParams<hip_bfloat16, hip_bfloat16, hip_bfloat16>;

template hipError_t SinglePrefillWithKVCacheDispatched<256, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, PosEncodingMode::kNone, 0, MaskMode::kNone, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    