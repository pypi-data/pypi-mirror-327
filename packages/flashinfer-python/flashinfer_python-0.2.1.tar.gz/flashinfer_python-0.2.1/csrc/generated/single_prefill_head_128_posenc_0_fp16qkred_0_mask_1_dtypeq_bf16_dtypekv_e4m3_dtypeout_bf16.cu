#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using ParamsT = SinglePrefillParams<hip_bfloat16, __hip_fp8_e4m3_fnuz, hip_bfloat16>;

template hipError_t SinglePrefillWithKVCacheDispatched<128, PosEncodingMode::kNone, 0, MaskMode::kCausal, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<128, PosEncodingMode::kNone, 0, MaskMode::kCausal, ComposedAttention<ParamsT, get_variant_code(
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false)>>(
    ParamsT params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    